#include "hip/hip_runtime.h"
#include "LevenshteinDistance.cuh"
#include "../CudaCheck.cuh"
#include <iostream>

void PrintArray(int* array, int m, int n, std::string s, std::string t)
{
	std::cout << "    ";
	for (int j = 0; j < n - 1; j++)
		std::cout << t[j] << " ";
	std::cout << std::endl;
	for (int i = 0; i < m; i++)
	{
		if (i == 0)
			std::cout << "  ";
		else
			std::cout << s[i - 1] << " ";

		for (int j = 0; j < n; j++)
			std::cout << array[i * n + j] << " ";
		std::cout << std::endl;
	}
	std::cout << std::endl << std::endl;
}

void PrintArray(char* array, int m, int n, std::string s, std::string t)
{
	std::cout << "    ";
	for (int j = 0; j < n - 1; j++)
		std::cout << t[j] << " ";
	std::cout << std::endl;
	for (int i = 0; i < m; i++)
	{
		if (i == 0)
			std::cout << "  ";
		else
			std::cout << s[i - 1] << " ";

		for (int j = 0; j < n; j++)
			std::cout << array[i * n + j] << " ";
		std::cout << std::endl;
	}
	std::cout << std::endl << std::endl;
}

int GPU::LevenshteinDistance::CalculateLevenshteinDistance(const std::string& sourceWord,
	const std::string& targetWord,
	std::string& transformation)
{
	SourceWordLength = static_cast<unsigned>(sourceWord.size());
	TargetWordLength = static_cast<unsigned>(targetWord.size());

	auto threadsInBlock = std::min(TargetWordLength + 1, THREADS_IN_ONE_BLOCK);
	auto blocksInGrid = (TargetWordLength + threadsInBlock) / threadsInBlock;

	int hostNextColumn = 0;

	// allocation
	CUDACHECK(hipMalloc((void**)&DeviceAlphabet, (AlphabetLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceSourceWord, (SourceWordLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceTargetWord, (TargetWordLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceX, AlphabetLength * (TargetWordLength + 1) * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&DeviceDistances, (SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&DeviceTransformations, (SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceNextColumn, sizeof(int)));

	// copying
	CUDACHECK(hipMemcpy(DeviceAlphabet, Alphabet, (AlphabetLength + 1) * sizeof(char), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(DeviceSourceWord, sourceWord.c_str(), (SourceWordLength + 1) * sizeof(char), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(DeviceTargetWord, targetWord.c_str(), (TargetWordLength + 1) * sizeof(char), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(DeviceNextColumn, &hostNextColumn, sizeof(int), hipMemcpyHostToDevice));

	PopulateDeviceX << <1u, AlphabetLength, (AlphabetLength + 1) * sizeof(char) >> > (
		DeviceX, 
		DeviceAlphabet,
		AlphabetLength, 
		DeviceTargetWord,
		TargetWordLength
	);
	
	CUDACHECK(hipPeekAtLastError());
	CUDACHECK(hipDeviceSynchronize());

	for (int i = 0; i < blocksInGrid; i++)
	{
		PopulateDeviceDistances << <1u, threadsInBlock, (threadsInBlock + SourceWordLength) * sizeof(char) >> >(
			DeviceDistances, 
			DeviceTransformations, 
			DeviceX, 
			DeviceSourceWord, 
			SourceWordLength, 
			DeviceTargetWord, 
			TargetWordLength, 
			WARP_SIZE, 
			DeviceNextColumn
		);

		CUDACHECK(hipPeekAtLastError());
		CUDACHECK(hipDeviceSynchronize());
	}

	int* hostDistances = new int[(SourceWordLength + 1) * (TargetWordLength + 1)];
	char* hostTransformations = new char[(SourceWordLength + 1) * (TargetWordLength + 1)];

	// copying
	CUDACHECK(hipMemcpy(hostDistances, DeviceDistances, (SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(int), hipMemcpyDeviceToHost));
	CUDACHECK(hipMemcpy(hostTransformations, DeviceTransformations, (SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(char), hipMemcpyDeviceToHost));

	int distance = hostDistances[SourceWordLength * (TargetWordLength + 1) + TargetWordLength];
	transformation = RetrieveTransformation(hostTransformations, SourceWordLength, TargetWordLength);

	PrintArray(hostDistances, SourceWordLength + 1, TargetWordLength + 1, sourceWord, targetWord);
	PrintArray(hostTransformations, SourceWordLength + 1, TargetWordLength + 1, sourceWord, targetWord);

	// free memory
	CUDACHECK(hipFree(DeviceAlphabet));
	CUDACHECK(hipFree(DeviceTransformations));
	CUDACHECK(hipFree(DeviceDistances));
	CUDACHECK(hipFree(DeviceX));
	CUDACHECK(hipFree(DeviceTargetWord));
	CUDACHECK(hipFree(DeviceSourceWord));
	CUDACHECK(hipFree(DeviceNextColumn));

	delete[] hostTransformations;
	delete[] hostDistances;

	return distance;
}

__device__ 
int GPU::ResolveTransformation(int s, int d, int i, char* transformation)
{
	int result = s;
	*transformation = SUBSTITUTE;

	if (d < result)
	{
		result = d;
		*transformation = DELETE;
	}

	if (i < result)
	{
		result = i;
		*transformation = INSERT;
	}

	return result;
}

std::string GPU::LevenshteinDistance::RetrieveTransformation(char* transformations, int m, int n)
{
	std::string transformation{};

	int i{ m }, j{ n };

	char current{};

	while (i != 0 || j != 0)
	{
		current = transformations[i * (n + 1) + j];
		transformation.push_back(current);

		if (current == DELETE) {
			i--;
		}
		else if (current == INSERT) {
			j--;
		}
		else {
			i--;
			j--;
		}
	}

	std::reverse(
		transformation.begin(),
		transformation.end()
	);

	return transformation;
}

__global__
void GPU::PopulateDeviceX(
	int* deviceX,
	char* deviceAlphabet,
	int alphabetLength,
	char* deviceTargetWord,
	int targetWordLength)
{
	int tid = threadIdx.x;

	extern __shared__ char sharedMemory[];

	sharedMemory[tid] = deviceAlphabet[tid];

	__syncthreads();

	int offset = tid * (targetWordLength + 1);

	deviceX[offset] = 0;

	for (int j = 1; j <= targetWordLength; ++j)
	{
		if (deviceTargetWord[j - 1] == sharedMemory[tid]) {
			deviceX[offset + j] = j;
		}
		else {
			deviceX[offset + j] = deviceX[offset + j - 1];
		}
	}
}

__global__ 
void GPU::PopulateDeviceDistances(
	int* deviceDistances,
	char* deviceTransformations,
	int* deviceX,
	char* deviceSourceWord,
	int sourceWordLenght,
	char* deviceTargetWord,
	int targetWordLength,
	int warpCount,
	int* deviceNextColumn)
{

	int tid = deviceNextColumn[0] + threadIdx.x;

	extern __shared__ char sharedMemory[];

	char* sharedSourceWord = sharedMemory + blockDim.x * sizeof(char);

	if (tid != 0 && tid <= targetWordLength) {
		sharedMemory[threadIdx.x] = deviceTargetWord[tid - 1];
	}

	int sharedSourceLength = (sourceWordLenght + blockDim.x) / blockDim.x;
	int sharedsourceStart = threadIdx.x * sharedSourceLength;

	for (int i = 0; i < sharedSourceLength && sharedsourceStart + i < sourceWordLenght; ++i) {
		sharedSourceWord[sharedsourceStart + i] = deviceSourceWord[sharedsourceStart + i];
	}

	__syncthreads();

	if (tid > targetWordLength) {
		return;
	}

	int aVar{}, bVar{}, cVar{}, dvar{ tid };
	char t{};

	deviceDistances[tid] = dvar;
	deviceTransformations[tid] = INSERT;

	for (int i = 1; i <= sourceWordLenght; ++i)
	{
		__syncthreads();

		int shuffledUp = __shfl_up(dvar, 1);

		if (tid != 0 && tid % warpCount == 0) {
			aVar = deviceDistances[(i - 1) * (targetWordLength + 1) + tid - 1];
		}
		else
		{
			if (tid != 0) {
				aVar = shuffledUp;
			}
		}

		char letter = sharedSourceWord[i - 1];
		int letterOffset = letter - 'a';

		int xVal = deviceX[letterOffset * (targetWordLength + 1) + tid];

		cVar = deviceDistances[(i - 1) * (targetWordLength + 1) + xVal - 1];

		__syncthreads();

		if (tid == 0)
		{
			dvar = i;
			t = DELETE;
		}

		else
		{

			if (sharedMemory[threadIdx.x] == letter)
			{
				dvar = aVar;
				t = SKIP;
			}
			else
			{
				bVar = dvar;

				if (xVal == 0) {
					dvar = 1 + GPU::ResolveTransformation(aVar, bVar, i + tid - 1, &t);
				}
				else {
					dvar = 1 + GPU::ResolveTransformation(aVar, bVar, cVar + (tid - 1 - xVal), &t);
				}
			}
		}

		deviceDistances[i * (targetWordLength + 1) + tid] = dvar;
		deviceTransformations[i * (targetWordLength + 1) + tid] = t;
	}

	if (threadIdx.x == 0) {
		deviceNextColumn[0] += blockDim.x;
	}
}