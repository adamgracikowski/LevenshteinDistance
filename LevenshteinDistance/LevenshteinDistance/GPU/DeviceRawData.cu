#include "DeviceRawData.cuh"

DeviceRawData::DeviceRawData(unsigned sourceWordLength, unsigned targetWordLength) :
	SourceWordLength{ sourceWordLength },
	TargetWordLength{ targetWordLength }
{
	CUDACHECK(hipMalloc((void**)&DeviceAlphabet, (AlphabetLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceSourceWord, (SourceWordLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceTargetWord, (TargetWordLength + 1) * sizeof(char)));
	CUDACHECK(hipMalloc((void**)&DeviceTransformations, (SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(char)));

	CUDACHECK(hipMalloc((void**)&DeviceX, AlphabetLength * (TargetWordLength + 1) * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&DeviceDistances, (SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&DeviceNextColumn, sizeof(int)));
}

DeviceRawData::~DeviceRawData()
{
	CUDACHECK(hipFree(DeviceAlphabet));
	CUDACHECK(hipFree(DeviceTransformations));
	CUDACHECK(hipFree(DeviceDistances));
	CUDACHECK(hipFree(DeviceX));
	CUDACHECK(hipFree(DeviceTargetWord));
	CUDACHECK(hipFree(DeviceSourceWord));
	CUDACHECK(hipFree(DeviceNextColumn));
}

void DeviceRawData::FromHost(const std::string& sourceWord, const std::string& targetWord, int* hostNextColumn)
{
	CUDACHECK(hipMemcpy(DeviceAlphabet, Alphabet, (AlphabetLength + 1) * sizeof(char), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(DeviceSourceWord, sourceWord.c_str(), (SourceWordLength + 1) * sizeof(char), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(DeviceTargetWord, targetWord.c_str(), (TargetWordLength + 1) * sizeof(char), hipMemcpyHostToDevice));

	CUDACHECK(hipMemcpy(DeviceNextColumn, hostNextColumn, sizeof(int), hipMemcpyHostToDevice));
}

void DeviceRawData::ToHost(int** hostDistances, char** hostTransformations)
{
	CUDACHECK(hipMemcpy(
		*hostDistances,
		DeviceDistances,
		(SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(int),
		hipMemcpyDeviceToHost
	));

	CUDACHECK(hipMemcpy(
		*hostTransformations,
		DeviceTransformations,
		(SourceWordLength + 1) * (TargetWordLength + 1) * sizeof(char),
		hipMemcpyDeviceToHost
	));
}