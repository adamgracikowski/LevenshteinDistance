#include "DeviceTimer.cuh"

namespace Timers
{
	void DeviceTimer::Start()
	{
		// Ensure old events are destroyed before creating new ones
		DestroyCudaEvents();
		InitCudaEvents();
		CUDACHECK(hipEventRecord(StartEvent));
	}

	void DeviceTimer::Stop() {
		if (StartEvent == nullptr || StopEvent == nullptr)
			return;

		CUDACHECK(hipEventRecord(StopEvent));
		CUDACHECK(hipEventSynchronize(StopEvent)); // Wait until the stop event is complete
		CUDACHECK(hipEventElapsedTime(&MilisecondsElapsed, StartEvent, StopEvent));

		TotalMilisecondsElapsed += MilisecondsElapsed;
	}

	float DeviceTimer::ElapsedMiliseconds() {
		return MilisecondsElapsed;
	}

	float DeviceTimer::TotalElapsedMiliseconds() {
		return TotalMilisecondsElapsed;
	}

	void DeviceTimer::Reset() {
		DestroyCudaEvents();
		MilisecondsElapsed = 0;
		TotalMilisecondsElapsed = 0;
	}

	void DeviceTimer::InitCudaEvents() {
		if (StartEvent == nullptr) {
			CUDACHECK(hipEventCreate(&StartEvent));
		}
		if (StopEvent == nullptr) {
			CUDACHECK(hipEventCreate(&StopEvent));
		}
	}

	void DeviceTimer::DestroyCudaEvents() {
		if (StartEvent != nullptr) {
			CUDACHECK(hipEventDestroy(StartEvent));
			StartEvent = nullptr;
		}
		if (StopEvent != nullptr) {
			CUDACHECK(hipEventDestroy(StopEvent));
			StopEvent = nullptr;
		}
	}

	DeviceTimer::~DeviceTimer() {
		Reset();
	}
}