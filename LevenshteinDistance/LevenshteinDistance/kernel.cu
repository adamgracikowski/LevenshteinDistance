﻿
#include "ProgramParameters.h"
#include "DataManager.h"
#include "Timers/TimerManager.h"

#include "CPU/LevenshteinDistance.h"
#include "GPU/LevenshteinDistance.cuh"

#include <iomanip>

#define SHOW_TABLES_SIZE 20

void DisplayProgramParameters(ProgramParameters parameters);

void DisplayWords(const std::string& source, const std::string& target);

void DisplaySummary(const std::string& computationMethod);

void TransformWords(std::string& sourceWord, std::string& targetWord);

int main(int argc, char* argv[])
{
	auto parameters = ParseProgramParameters(argc, argv);

	// Check if the parameters are successfully parsed
	if (!parameters.Success) {
		return 1;
	}

	DisplayProgramParameters(parameters);

	try {
		auto dataManager = DataManager{};

		auto loaded = dataManager.LoadDataFromInputFile(parameters.DataFormat, parameters.InputFile);
		auto& sourceWord = loaded.first;
		auto& targetWord = loaded.second;

		// Transform both source and target words to lowercase before computation
		TransformWords(sourceWord, targetWord);

		DisplayWords(sourceWord, targetWord);

		// Check if the word lengths are small enough to display tables
		auto showTables = sourceWord.length() <= SHOW_TABLES_SIZE && targetWord.length() <= SHOW_TABLES_SIZE;

		std::string transformation{};

		if (parameters.ComputationMethod == CPU_COMPUTATION_METHOD) {
			auto lev = CPU::LevenshteinDistance{};
			lev.CalculateLevenshteinDistance(sourceWord, targetWord, transformation, showTables);
		}
		else {
			auto lev = GPU::LevenshteinDistance{};
			lev.CalculateLevenshteinDistance(sourceWord, targetWord, transformation, showTables);
		}

		std::cout << transformation << std::endl;

		dataManager.SaveDataToOutputFile(parameters.OutputFile, TXT_FORMAT, transformation);

		DisplaySummary(parameters.ComputationMethod);
	}
	catch (const std::exception& e) {
		// Handle any exceptions that occur during computation
		std::cerr << e.what() << std::endl;
	}

	return 0;
}

void DisplayProgramParameters(ProgramParameters parameters)
{
	std::cout << std::setw(25) << std::left << "Data format: "
		<< parameters.DataFormat << std::endl;
	std::cout << std::setw(25) << std::left << "Computation method: "
		<< parameters.ComputationMethod << std::endl << std::endl;
}

void DisplayWords(const std::string& source, const std::string& target)
{
	const auto width = 50;

	auto m = source.length();
	auto n = target.length();

	if (m < width) {
		std::cout << std::setw(25) << std::left << "Source word: "
			<< source << std::endl;
	}
	else {
		std::cout << std::setw(25) << std::left << "Source word length: "
			<< source.length() << std::endl;
	}

	if (n < width) {
		std::cout << std::setw(25) << std::left << "Target word: "
			<< target << std::endl;
	}
	else {
		std::cout << std::setw(25) << std::left << "Target word length: "
			<< target.length() << std::endl;
	}

	std::cout << std::endl;
}

void DisplaySummary(const std::string& computationMethod)
{
	const int width{ 40 };
	auto& timerManager = Timers::TimerManager::GetInstance();

	auto totalExecutionTime{ 0.0f };
	auto totalAlgorithmTime{ 0.0f };

	auto displayTime = [&](const std::string& title, const float elapsed)
		{
			totalExecutionTime += elapsed;
			std::cout << std::setw(width) << std::left << title << elapsed << " ms." << std::endl;
		};

	std::cout << std::endl;
	displayTime("Loading data from the input file: ", timerManager.LoadDataFromInputFileTimer.TotalElapsedMiliseconds());

	if (computationMethod == CPU_COMPUTATION_METHOD) {
		displayTime("Finding distance: ", timerManager.FindDistanceTimer.TotalElapsedMiliseconds());
		totalAlgorithmTime += timerManager.FindDistanceTimer.TotalElapsedMiliseconds();

		displayTime("Retrieving transformation: ", timerManager.RetrieveTransformationTimer.TotalElapsedMiliseconds());
		totalAlgorithmTime += timerManager.RetrieveTransformationTimer.TotalElapsedMiliseconds();
	}
	else {
		displayTime("Host to device transfer: ", timerManager.Host2DeviceDataTransferTimer.TotalElapsedMiliseconds());
		totalAlgorithmTime += timerManager.Host2DeviceDataTransferTimer.TotalElapsedMiliseconds();

		displayTime("Populating X: ", timerManager.PopulateDeviceXTimer.TotalElapsedMiliseconds());
		totalAlgorithmTime += timerManager.PopulateDeviceXTimer.TotalElapsedMiliseconds();

		displayTime("Populating distances: ", timerManager.PopulateDeviceDistancesTimer.TotalElapsedMiliseconds());
		totalAlgorithmTime += timerManager.PopulateDeviceDistancesTimer.TotalElapsedMiliseconds();

		displayTime("Device to host transfer: ", timerManager.Device2HostDataTransferTimer.TotalElapsedMiliseconds());
		totalAlgorithmTime += timerManager.Device2HostDataTransferTimer.TotalElapsedMiliseconds();
	}

	std::cout << std::setw(40) << std::left << "Saving data to output file time: "
		<< timerManager.SaveDataToOutputFileTimer.TotalElapsedMiliseconds() << " ms." << std::endl << std::endl;

	std::cout << std::setw(width) << std::left << "Total algorithm execution: " << totalExecutionTime << " ms." << std::endl;
	std::cout << std::setw(width) << std::left << "Total execution: " << totalExecutionTime << " ms." << std::endl;
}

void TransformWords(std::string& sourceWord, std::string& targetWord)
{
	std::transform(
		sourceWord.begin(),
		sourceWord.end(),
		sourceWord.begin(),
		[](unsigned char c) {
			return tolower(c);
		}
	);

	std::transform(
		targetWord.begin(),
		targetWord.end(),
		targetWord.begin(),
		[](unsigned char c) {
			return tolower(c);
		}
	);
}